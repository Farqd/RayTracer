
#include <hip/hip_runtime.h>
#include<cstdio>

struct Point
{
	double x; 
	double y; 
	double z;
};
struct pairPd
{
	Point first;
	double second;
};
struct pairbp
{
	bool first;
	pairPd second;
};
struct RGB
{
   unsigned char r;
   unsigned char g;
   unsigned char b;
};



struct Sphere
{
	Point center;
	double radius;
	RGB color;
};

struct Segment
{
	Point a;
	Point b;
};

extern "C" {
__device__
bool isCloseToZero(double x)
{
    return abs(x) < 0.00000001;
}
__device__
inline RGB operator*(RGB rgb, double const& times)
{
  rgb.r *= times;
  rgb.g *= times;
  rgb.b *= times;
  
  return rgb;
}
__device__
pairbp intersection(Segment segment, Sphere sphere)
{
	double x0 = segment.a.x;
	double y0 = segment.a.y;
	double z0 = segment.a.z;

	double x1 = segment.b.x;
	double y1 = segment.b.y;
	double z1 = segment.b.z;

	double dx = x1 - x0;
	double dy = y1 - y0;
	double dz = z1 - z0;

	double cx = sphere.center.x;
	double cy = sphere.center.y;
	double cz = sphere.center.z;
	
	double a = dx*dx + dy*dy + dz*dz;
	double b = 2*dx*(x0-cx) +  2*dy*(y0-cy) +  2*dz*(z0-cz);
	double c = cx*cx + cy*cy + cz*cz + x0*x0 + y0*y0 + z0*z0 -2*(cx*x0 + cy*y0 + cz*z0) - sphere.radius * sphere.radius;

	double discriminant = b*b - 4*a*c;
	if(!isCloseToZero(discriminant) && discriminant < 0)
		return {false, {} };

	double t = (-b - sqrt(discriminant)) / (2*a);
	if(t < 0)
		return {false, {} };
	return {true, {{x0 + t*dx, y0 + t*dy, z0 + t*dz}, t }};

}



__device__
double vectorlen(Point const& vec)
{
	return sqrt(vec.x*vec.x + vec.y*vec.y + vec.z*vec.z);
}
__device__
double dotProduct(Point const&a, Point const& b)
{
	return a.x*b.x + a.y*b.y + a.z*b.z;
}
__device__
bool pointInShadow(Point const& point, Point const& light, Sphere const& sphere)
{
	Segment seg = {point, light};
	return intersection(seg, sphere).first;
}
__device__
void normalize(Point& vec)
{
	double len = vectorlen(vec);
	vec.x = vec.x / len;
	vec.y = vec.y / len;
	vec.z = vec.z / len;
}

__device__
void processPixelOnBackground(RGB* bitmap, Sphere* spheres, Point const& pixel, int spheresNum, int imageY, int imageZ, Point const& observer, Point const& light, RGB const& background)
{
	
	int idx = ((blockIdx.x * blockDim.x) + threadIdx.x)*imageZ*2 + (blockIdx.y * blockDim.y) + threadIdx.y;
	//int idx = (pixel.y + imageY)*imageZ*2 +  pixel.z + imageZ; 

	if(pixel.y - observer.y >= 0)
	{
		//bitmap[pixel.y + imageY][pixel.z + imageZ] = {30, 30, 30};

		bitmap[idx].r = 30;
		bitmap[idx].g = 30;
		bitmap[idx].b = 30;
		return;
	}

	Point pointOnFloor;
	pointOnFloor.y = -400;
	double times = - 400 / (pixel.y - observer.y);
	
	pointOnFloor.x = (pixel.x - observer.x) * times;
	pointOnFloor.z = (pixel.z - observer.z) * times;

	Segment seg = {pointOnFloor, light};

	bool isInShadow = false;
	//for(auto const& sphere : spheres)
	for (int i=0; i<spheresNum; ++i)	
	{
		Sphere sphere = spheres[i];
		if(intersection(seg, sphere).first)
		{
			isInShadow = true; 
			break;
		}
	}

	if(isInShadow)
	{
		//bitmap[pixel.y + imageY][pixel.z + imageZ] = { uint8_t(background.r/2), uint8_t(background.g/2), uint8_t(background.b/2)};
		
		bitmap[idx].r = background.r/2;
		bitmap[idx].g = background.g/2;
		bitmap[idx].b = background.b/2;
	}

	else
	{
		bitmap[idx] = background;
	}

}

__global__
void processPixel(Sphere* spheres, int spheresNum, RGB* bitmap, int imageX, int imageY, int imageZ, int antiAliasing, double diffuseCoefficient, double ambientCoefficient, double observerX, double observerY, double observerZ, double lX, double lY, double lZ, unsigned char R, unsigned char G, unsigned char B) {
	    

	Point const observer = {observerX, observerY, observerZ};
	Point const light = {lX, lY, lZ};
	RGB background = {R, G, B};

	//int idx = (pixel.y + imageY)*imageZ*2 +  pixel.z + imageZ; 
	int thidX = (blockIdx.x * blockDim.x) + threadIdx.x;
	int thidY = (blockIdx.y * blockDim.y) + threadIdx.y;
    
	
    if (thidX<2*imageY && thidY<2*imageZ)
    {

	Point point{imageX, ((double)(thidX-imageY))/antiAliasing, ((double)(thidY-imageZ))/antiAliasing};
		
	//Point point{imageX, (thidX-imageY), (thidY-imageZ)};
    Segment seg{observer, point };
	//std::vector<std::pair<std::pair<Point, double>, size_t>> distanceIndex;
	Point dIff;
	double dIfs;
	size_t dIs;
	
	bool intersected = false;
	//for(size_t i = 0; i<spheres.size(); i++)
	
	for (int i=0; i<spheresNum; ++i)	
	{
		Sphere const& sphere = spheres[i];
		pairbp const& res = intersection(seg, sphere);
		if(res.first)
			{
				if (!intersected || res.second.second < dIfs) {dIff = res.second.first; dIfs = res.second.second; dIs = i;}
				intersected = true;
				
				//distanceIndex.push_back({ {res.second}, i});
			}
	}
	
	//if(!distanceIndex.empty())
	if (intersected)	
	{
		//std::sort(distanceIndex.begin(), distanceIndex.end(),
		//	[](std::pair<std::pair<Point, double>, int> const& a, std::pair<std::pair<Point, double>, int> const& b)
		//	{ return a.first.second < b.first.second; } );

		Point const& pointOnSphere =  dIff; //distanceIndex[0].first.first;
		Point const& center = spheres[dIs].center; //spheres[distanceIndex[0].second].center;
		double radius = spheres[dIs].radius; //spheres[distanceIndex[0].second].radius;
		RGB rgb = spheres[dIs].color; //spheres[distanceIndex[0].second].color;

		bool isInShadow = false;
		//for(size_t i=0; i<spheres.size(); i++)
		for (int i=0; i<spheresNum; ++i)		
		{
			//if(i != distanceIndex[0].second && pointInShadow(pointOnSphere, light, spheres[i]))
			if (i!=dIs && pointInShadow(pointOnSphere, light, spheres[i]))			
			{
				isInShadow = true; 
				break;
			}
		}
		int idx = thidX*imageZ*2 + thidY;
		if(isInShadow)
		{
			bitmap[idx] = rgb *ambientCoefficient;
		}
		else
		{
			Point normalVector = {(pointOnSphere.x - center.x)/radius, (pointOnSphere.y - center.y)/radius, (pointOnSphere.z - center.z)/radius};
			Point unitVec = {light.x - pointOnSphere.x, light.y - pointOnSphere.y, light.z - pointOnSphere.z};
			normalize(unitVec);
			double dot = dotProduct(normalVector, unitVec);

			bitmap[idx] = rgb* (max(0.0, diffuseCoefficient * dot) + ambientCoefficient);
		}
	}
	else
		processPixelOnBackground(bitmap, spheres, point, spheresNum, imageY, imageZ, observer, light, background);    
    }
}
}

